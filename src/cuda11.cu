
#include <hip/hip_runtime.h>
__global__ void _add_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi+yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void add_32_11(int n, float *x, float *y, float *z) {
    _add_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _add_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi+yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void add_64_11(int n, double *x, double *y, double *z) {
    _add_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _sub_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi-yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void sub_32_11(int n, float *x, float *y, float *z) {
    _sub_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _sub_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi-yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void sub_64_11(int n, double *x, double *y, double *z) {
    _sub_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _mul_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi*yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void mul_32_11(int n, float *x, float *y, float *z) {
    _mul_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _mul_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi*yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void mul_64_11(int n, double *x, double *y, double *z) {
    _mul_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _div_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi/yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void div_32_11(int n, float *x, float *y, float *z) {
    _div_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _div_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi/yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void div_64_11(int n, double *x, double *y, double *z) {
    _div_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _pow_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = pow(xi,yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void pow_32_11(int n, float *x, float *y, float *z) {
    _pow_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _pow_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = pow(xi,yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void pow_64_11(int n, double *x, double *y, double *z) {
    _pow_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _max_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (xi>yi?xi:yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void max_32_11(int n, float *x, float *y, float *z) {
    _max_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _max_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (xi>yi?xi:yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void max_64_11(int n, double *x, double *y, double *z) {
    _max_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _min_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (xi<yi?xi:yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void min_32_11(int n, float *x, float *y, float *z) {
    _min_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _min_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (xi<yi?xi:yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void min_64_11(int n, double *x, double *y, double *z) {
    _min_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _eq_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi==yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void eq_32_11(int n, float *x, float *y, float *z) {
    _eq_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _eq_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi==yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void eq_64_11(int n, double *x, double *y, double *z) {
    _eq_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _ne_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi!=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void ne_32_11(int n, float *x, float *y, float *z) {
    _ne_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _ne_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi!=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void ne_64_11(int n, double *x, double *y, double *z) {
    _ne_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _gt_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi>yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void gt_32_11(int n, float *x, float *y, float *z) {
    _gt_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _gt_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi>yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void gt_64_11(int n, double *x, double *y, double *z) {
    _gt_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _ge_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi>=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void ge_32_11(int n, float *x, float *y, float *z) {
    _ge_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _ge_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi>=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void ge_64_11(int n, double *x, double *y, double *z) {
    _ge_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _lt_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi<yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void lt_32_11(int n, float *x, float *y, float *z) {
    _lt_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _lt_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi<yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void lt_64_11(int n, double *x, double *y, double *z) {
    _lt_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _le_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = xi<=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void le_32_11(int n, float *x, float *y, float *z) {
    _le_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _le_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = xi<=yi;
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void le_64_11(int n, double *x, double *y, double *z) {
    _le_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _invxback_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (-xi*yi*yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void invxback_32_11(int n, float *x, float *y, float *z) {
    _invxback_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _invxback_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (-xi*yi*yi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void invxback_64_11(int n, double *x, double *y, double *z) {
    _invxback_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _reluback_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (yi>0?xi:0);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void reluback_32_11(int n, float *x, float *y, float *z) {
    _reluback_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _reluback_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (yi>0?xi:0);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void reluback_64_11(int n, double *x, double *y, double *z) {
    _reluback_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _sigmback_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (xi*yi*(1-yi));
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void sigmback_32_11(int n, float *x, float *y, float *z) {
    _sigmback_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _sigmback_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (xi*yi*(1-yi));
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void sigmback_64_11(int n, double *x, double *y, double *z) {
    _sigmback_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _tanhback_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = (xi*(1-yi*yi));
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void tanhback_32_11(int n, float *x, float *y, float *z) {
    _tanhback_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _tanhback_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = (xi*(1-yi*yi));
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void tanhback_64_11(int n, double *x, double *y, double *z) {
    _tanhback_64_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _rpow_32_11(int n, float *x, float *y, float *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    float xi=x[i];
    float yi=y[i];
    z[i] = pow(yi,xi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void rpow_32_11(int n, float *x, float *y, float *z) {
    _rpow_32_11<<<256,256>>>(n,x,y,z);
  }    
}
__global__ void _rpow_64_11(int n, double *x, double *y, double *z) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    double xi=x[i];
    double yi=y[i];
    z[i] = pow(yi,xi);
    i += blockDim.x * gridDim.x;
  }
}
extern "C" {
  void rpow_64_11(int n, double *x, double *y, double *z) {
    _rpow_64_11<<<256,256>>>(n,x,y,z);
  }    
}
